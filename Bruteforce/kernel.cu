#include "hip/hip_runtime.h"
﻿#include ""

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <sstream>
#include <iostream>
#include <iomanip>
#include <algorithm>
#include <vector>

uint8_t array1[] = {
        0x21, 0x46, 0x93, 0xA8, 0x02, 0x48, 0xB3, 0x49,
        0x9E, 0xA7, 0xD3, 0x9E, 0xA7, 0xD3, 0x8E, 0xA3,
        0x1D, 0xA3, 0x7D, 0xAF, 0xD6, 0xB5, 0xE5, 0xBC,
        0xDB, 0xDE, 0xF7, 0x5B, 0xDA, 0x6F, 0xB7, 0x8C,
        0x17, 0x7B, 0xD8, 0x5F, 0x31, 0xA6, 0xA5
};

uint8_t array2[] = {
    0x21, 0x46, 0x58, 0x57, 0xFE, 0xBB, 0x02, 0x56,
    0x14, 0x41, 0x82, 0x0A, 0x28, 0x28, 0x2A, 0xA8,
    0xA8, 0xA8, 0xA4, 0x4A, 0x21, 0x42, 0x63, 0x1B,
    0xD0, 0xD0, 0xD1, 0x21, 0x42, 0x0D, 0xA1, 0x3D,
    0x57, 0x91, 0xE2, 0x37, 0x9D, 0x76, 0xB5
};

__device__ int min3(int a, int b, int c) {
    if (a < b && a < c) return a;
    if (b < a && b < c) return b;
    return c;
}

// Function to find Levenshtein Distance between string1 and string2
__device__ int countLevenshteinDistance(const char* str1, const char* str2) {
    const int len1 = 78, len2 = 78;
    int cost[len1 + 1][len2 + 1];

    // Initializing cost array
    for (int i = 0; i <= len1; i++) cost[i][0] = i;
    for (int j = 0; j <= len2; j++) cost[0][j] = j;

    // Calculating costs
    for (int i = 1; i <= len1; i++) {
        for (int j = 1; j <= len2; j++) {
            int costOfSubstitution = (str1[i - 1] == str2[j - 1]) ? 0 : 1;
            cost[i][j] = min3(
                cost[i - 1][j] + 1,                 // Deletion
                cost[i][j - 1] + 1,                 // Insertion
                cost[i - 1][j - 1] + costOfSubstitution  // Substitution
            );
        }
    }

    return cost[len1][len2];
}

//Remember to reserve 2*size+1 bytes for the output!!!
__device__ void arrToHex(const uint8_t* arr, size_t size, char* output) {
    const char* cyfryHex = "0123456789ABCDEF";
    for (size_t i = 0; i < size; ++i) {
        output[i * 2] = cyfryHex[(arr[i] >> 4) & 0xF];
        output[i * 2 + 1] = cyfryHex[arr[i] & 0xF];
    }
    output[size * 2] = '\0'; //The end of the string
}

__device__ void numberToByteArr(uint32_t number, uint8_t output[4]) {
    for (int i = 0; i < 2; ++i) {
        output[i] = (number >> (i * 8)) & 0xFF;
    }
}

__device__ void codeXOR(const uint8_t* arr, const uint8_t* key, uint8_t output[39]) {
    for (size_t i = 0; i < 39; ++i) {
        output[i] = arr[i] ^ key[i % 2];
    }
}

__global__ void decrypt(uint8_t* arr1, uint8_t* arr2, uint8_t* out_score) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    uint8_t key1[2];
    uint8_t key2[2];
    numberToByteArr(i, key1);
    numberToByteArr(j, key2);
    uint8_t arr1XOR[39];
    uint8_t arr2XOR[39];
    codeXOR(arr1, key1, arr1XOR);
    codeXOR(arr2, key2, arr2XOR);
    char arr1XORstring[79];
    char arr2XORstring[79];
    arrToHex(arr1XOR, 39, arr1XORstring);
    arrToHex(arr2XOR, 39, arr2XORstring);
    int score = countLevenshteinDistance(arr1XORstring, arr2XORstring);
    if (score < 20) {
        out_score[0] = score;
        out_score[1] = key1[0];
        out_score[2] = key1[1];
        out_score[3] = key1[2];
        out_score[4] = key1[3];
        out_score[5] = key2[0];
        out_score[6] = key2[1];
        out_score[7] = key2[2];
        out_score[8] = key2[3];

    }
}


int main() {
    printf("Start\n");

    // Rozpoczęcie pomiaru czasu
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    //Alokacja pamięci na urządzeniu
    size_t size = 39 * sizeof(uint8_t);
    uint8_t* d_array1;
    hipMalloc(&d_array1, size);
    uint8_t* d_array2;
    hipMalloc(&d_array2, size);

    uint8_t score[9];
    size_t size_u = 9 * sizeof(uint8_t);   //  {calculated_distance, 1p1b, 1p2b, 2p1b, 2p2b}
    uint8_t* d_score;
    hipMalloc(&d_score, size_u);


    //Kopiowanie zmiennych do GPU
    hipMemcpy(d_array1, array1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_array2, array2, size, hipMemcpyHostToDevice);

    // Uruchomienie kernela
    dim3 thredsPerBlock(32, 32, 1);
    dim3 numBlocks(2048, 2048, 1);
    decrypt << <numBlocks, thredsPerBlock >> > (d_array1, d_array2, d_score);

    hipDeviceSynchronize();

    hipMemcpy(score, d_score, size_u, hipMemcpyDeviceToHost);

    // Zakończenie pomiaru czasu
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Czas trwania: %f ms\n", time);
    printf("Najmniejsza odleglosc: %d\n", score[0]);
    printf("Klucz 1: %02X%02X%02X%02X\nKlucz 2: %02X%02X%02X%02X\n", score[1], score[2], score[3], score[4], score[5], score[6], score[7], score[8]);

// Zwolnienie pamięci
    hipFree(d_array1);
    hipFree(d_array2);
    hipFree(d_score);

    return 0;
}